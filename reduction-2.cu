#include <iostream>
#include<hip/hip_runtime.h>
#include <chrono>
#include <numeric> 

// REDUCTION 2 – Sequence Addressing
__global__ void reduce2(int *g_in_data, int *g_out_data){
    extern __shared__ int sdata[];  // stored in the shared memory

    // Each thread loading one element from global onto shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    sdata[tid] = g_in_data[i];
    __syncthreads();

    // Reduction method -- occurs in shared memory
    for(unsigned int s = blockDim.x/2; s > 0; s >>= 1){
        // check out the reverse loop above
        if (tid < s){   // then, we check threadID to do our computation
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    if (tid == 0){
        g_out_data[blockIdx.x] = sdata[0];
    }
}

// I hope to use this main file for all of the reduction files
int main(){
    int n = 1 << 22; // Increase to about 4M elements
    size_t bytes = n * sizeof(int);

    // Host/CPU arrays
    int *host_input_data = new int[n];
    int *host_output_data = new int[(n + 255) / 256]; // to have sufficient size for output array

    // Device/GPU arrays
    int *dev_input_data, *dev_output_data;

    // Init data
    srand(42); // Fixed seed
    for (int i = 0; i < n; i++){
        host_input_data[i] = rand() % 100;
    }

    // Allocating memory on GPU for device arrays
    hipMalloc(&dev_input_data, bytes);
    hipMalloc(&dev_output_data, (n + 255) / 256 * sizeof(int));

    // Copying our data onto the device (GPU)
    hipMemcpy(dev_input_data, host_input_data, bytes, hipMemcpyHostToDevice);

    int blockSize = 256; // number of threads per block

    auto start = std::chrono::high_resolution_clock::now(); // start timer

    // Launch Kernel and Synchronize threads
    int num_blocks = (n + blockSize - 1) / blockSize;
    hipError_t err;
    reduce2<<<num_blocks, blockSize, blockSize * sizeof(int)>>>(dev_input_data, dev_output_data);
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
    }
    hipDeviceSynchronize();

    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start).count() / 1000.0; // duration in milliseconds with three decimal points

    // Copying data back to the host (CPU)
    hipMemcpy(host_output_data, dev_output_data, (n + 255) / 256 * sizeof(int), hipMemcpyDeviceToHost);

    // Final reduction on the host
    int finalResult = host_output_data[0];
    for (int i = 1; i < (n + 255) / 256; ++i) {
        finalResult += host_output_data[i];
    }

    // CPU Summation for verification
    int cpuResult = std::accumulate(host_input_data, host_input_data + n, 0);
    if (cpuResult == finalResult) {
        std::cout << "Verification successful: GPU result matches CPU result.\n";
        std::cout << "GPU Result: " << finalResult << ", CPU Result: " << cpuResult << std::endl;
    } else {
        std::cout << "Verification failed: GPU result (" << finalResult << ") does not match CPU result (" << cpuResult << ").\n";
        std::cout << "GPU Result: " << finalResult << ", CPU Result: " << cpuResult << std::endl;
    }

    double bandwidth = (duration > 0) ? (bytes / duration / 1e6) : 0; // computed in GB/s, handling zero duration
    std::cout << "Reduced result: " << finalResult << std::endl;
    std::cout << "Time elapsed: " << duration << " ms" << std::endl;
    std::cout << "Effective bandwidth: " << bandwidth << " GB/s" << std::endl;

    // Freeing memory
    hipFree(dev_input_data);
    hipFree(dev_output_data);
    delete[] host_input_data;
    delete[] host_output_data;
}